//CS4370
//Parallel Programming for Many-Core GPUs
//Avery Colley
//Meilin Liu
//12/12/24
//Programming Assignment #1 - Basic Matrix Multiplication
//Compiling command: nvcc Colley_Project1Task2.cu -o matrixMult

#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

int verify(int *a, int *b, int size) {
	for(int i = 0; i < size; i++) {
		if(a[i] != b[i]) {
			printf("TEST FAILED\n");
			return 0;
		}
	}
	printf("TEST PASSED\n");
	return 1;
}

void cpuMatrixMulti(int *M, int *N, int *P, int width) {
	for(int i = 0; i < width; ++i) {
		for(int j = 0; j < width; ++j) {
			int sum = 0;
			for(int k = 0; k < width; ++k) {
				int a = M[i * width + k];
				int b = N[k * width + j];
				sum += a * b;
			}
			P[i * width + j] = sum;
		}
	}
}

__global__ void gpuMatrixMulti(int *M, int *N, int *P, int width) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if((row < width) && (col < width)) {
		int pValue = 0;
		for(int k = 0; k < width; ++k) {
			pValue += M[row * width + k] * N[k * width + col];
		}
		__syncthreads();
		P[row * width + col] = pValue;
	}
}

int main() {
	int *A;
	int *B;
	int *C;
	int N = 1024;
	A = (int*) malloc(sizeof(int) * N * N);
	B = (int*) malloc(sizeof(int) * N * N);
	C = (int*) malloc(sizeof(int) * N * N);
	int init = 1325;
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			init = 3125 * init % 65536;
			A[(i * N) + j] = (init - 32768) / 6553;
			B[(i * N) + j] = init % 1000;
			C[(i * N) + j] = 0;
		}
	}
	printf("Input matrices have %d elements per row\n", N);

	int *a;
	int *b;
	int *c;
	int *result = (int*) malloc(sizeof(int) * N *N);

	hipMalloc((void**)(&a), sizeof(int) * N * N);
	hipMalloc((void**)(&b), sizeof(int) * N * N);
	hipMalloc((void**)(&c), sizeof(int) * N * N);

	hipMemcpy(a, A, sizeof(int) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(b, B, sizeof(int) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(c, C, sizeof(int) * N * N, hipMemcpyHostToDevice);

	const int BLOCK_SIZE = 16;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil((double) N / dimBlock.x), ceil((double) N / dimBlock.y));
	printf("Using %d thread blocks of %dx%d threads each\n", dimGrid.x * dimGrid.y, dimBlock.x, dimBlock.y);

	double cpuStartTime;
	double cpuTimeDifference;
	cpuStartTime = (double) clock();
	cpuMatrixMulti(A, B, C, N);
	cpuTimeDifference = ((double) clock() - cpuStartTime) / CLOCKS_PER_SEC;

	double gpuStartTime;
	double gpuTimeDifference;
	gpuStartTime = (double) clock();
	gpuMatrixMulti<<<dimGrid, dimBlock>>>(a, b, c, N);
	gpuTimeDifference = ((double) clock() - gpuStartTime) / CLOCKS_PER_SEC;

	hipMemcpy(result, c, sizeof(int) * N * N, hipMemcpyDeviceToHost);
	verify(C, result, N * N);
	printf("CPU runtime: %f seconds\n", cpuTimeDifference);
	printf("GPU runtime: %f seconds\n", gpuTimeDifference);

	printf("First row of CPU result matrix: [");
	for(int i = 0; i < N; i++) {
		printf("%d", C[i]);
		printf(" ");
	}
	printf("]\n");

	printf("First row of GPU result matrix: [");
	for(int i = 0; i < N; i++) {
		printf("%d", result[i]);
		printf(" ");
	}
	printf("]\n");

	free(A);
	free(B);
	free(C);
	free(result);
	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}