//CS4370
//Parallel Programming for Many-Core GPUs
//Avery Colley
//Meilin Liu
//12/11/24
//Programming Assignment #3Task1 - Work Efficient Parallel Reduction
//Compiling command: nvcc Colley_Project3Task1.cu -o sum

#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

int cpuSumReduction(int* x, int N) {
	for(int i = 1; i < N; i++) {
		x[0] = x[0] + x[i];
	}
	int overallSum = x[0];
	return overallSum;
}

int verify(int a, int b) {
	if(a == b) {
		printf("TEST PASSED\n");
		return 1;
	}
	printf("TEST FAILED\n");
	return 0;
}

void makeBlockArray(int* input, int* output, int N) {
	int j = 0;
	for(int i = 0; i < N; i = i + 1024) {
		output[j] = input[i];
		j++;
	}
}


__global__ void gpuSumReduction(unsigned int* input) {
	__shared__ int partialSum[2 * 512];

	unsigned int tx = threadIdx.x;
	unsigned int start = 2 * blockIdx.x * blockDim.x;
	partialSum[tx] = input[start + tx];
	partialSum[blockDim.x + tx] = input[start + blockDim.x + tx];

	for(unsigned int stride = blockDim.x; stride > 0; stride /= 2) {
		if(tx < stride) {
			partialSum[tx] += partialSum[tx + stride];
		}
		__syncthreads();
	}
	__syncthreads();

	if(tx == 0) {
		input[2 * 512 * blockIdx.x] = partialSum[tx];
	}
}

int main() {
	int* A;
	int N = 131072;
	A = (int*) malloc(sizeof(int) * N);
	int init = 1325;
	for(int i = 0; i < N; i++) {
		init = (3125 * init) % 6553;
		A[i] = (init - 1000) % 97;
	}
	int firstInput = A[0];
	printf("Input array has %d elements\n", N);
	dim3 dimBlock(512);
	dim3 dimGrid(ceil((double) N / (2 * dimBlock.x)));
	printf("Using %d thread blocks of %d threads each\n", dimGrid.x, dimBlock.x);

	int* result = (int*) malloc(sizeof(int) * N);
	int* B = (int*) malloc(sizeof(int) * (N / (dimBlock.x * 2)));
	int* gpuFinal = (int*) malloc(sizeof(int) * (N / (dimBlock.x * 2)));

	unsigned int* a;
	unsigned int* b;
	hipMalloc((void**)(&a), sizeof(int) * N);
	hipMalloc((void**)(&b), sizeof(int) * (N / (dimBlock.x * 2)));
	hipMemcpy(a, A, sizeof(int) * N, hipMemcpyHostToDevice);

	double cpuStartTime;
	double cpuTimeDifference;
	cpuStartTime = (double) clock();
	int cpuFinal = cpuSumReduction(A, N);
	cpuTimeDifference = ((double) clock() - cpuStartTime) / CLOCKS_PER_SEC;

	double gpuComputationTime = 0;
	double gpuStartTime;
	double gpuTimeDifference;
	gpuStartTime = (double) clock();
	gpuSumReduction<<<dimGrid, dimBlock.x>>>(a);
	gpuTimeDifference = ((double) clock() - gpuStartTime) / CLOCKS_PER_SEC;
	gpuComputationTime += gpuTimeDifference;
	hipMemcpy(result, a, sizeof(int) * N, hipMemcpyDeviceToHost);
	makeBlockArray(result, B, N);
	hipMemcpy(b, B, sizeof(int) * (N / (dimBlock.x * 2)), hipMemcpyHostToDevice);
	gpuStartTime = (double) clock();
	gpuSumReduction<<<1, dimBlock.x>>>(b);
	gpuTimeDifference = ((double) clock() - gpuStartTime) / CLOCKS_PER_SEC;
	gpuComputationTime += gpuTimeDifference;

	hipMemcpy(gpuFinal, b, sizeof(int) * (N / (dimBlock.x * 2)), hipMemcpyDeviceToHost);

	verify(cpuFinal, gpuFinal[0]);
	printf("CPU runtime: %f seconds\n", cpuTimeDifference);
	printf("GPU runtime: %f seconds\n", gpuComputationTime);

	printf("First 20 elements of the input array: %d ", firstInput);
	for(int i = 1; i < 20; i++) {
		printf("%d", A[i]);
		printf("  ");
	}
	printf("\n");

	printf("First 20 elements of the CPU parallel reduction array: ");
	for(int i = 0; i < 20; i++) {
		printf("%d", A[i]);
		printf("  ");
	}
	printf("\n");

	printf("First 20 elements of the GPU parallel reduction array: ");
	for(int i = 0; i < 20; i++) {
		printf("%d", gpuFinal[i]);
		printf("  ");
	}
	printf("\n");

	free(A);
	free(B);
	free(result);
	free(gpuFinal);
	hipFree(a);
	hipFree(b);
	return 0;
}