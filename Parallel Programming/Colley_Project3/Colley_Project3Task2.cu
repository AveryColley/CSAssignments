//CS4370
//Parallel Programming for Many-Core GPUs
//Avery Colley
//Meilin Liu
//12/11/24
//Programming Assignment #3Task2 - Work Efficient Parallel Prefix Sum
//Compiling command: nvcc Colley_Project3Task2.cu -o scan

#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

void cpuPrefixScan(int* y, int* x, int N) {
	y[0] = x[0];
	for(int i = 1; i < N; i++) {
		y[i] = y[i - 1] + x[i];
	}
}

int verify(int *a, int *b, int size) {
	for(int i = 0; i < size; i++) {
		if(a[i] != b[i]) {
			printf("TEST FAILED at position %d\n", i);
			return 0;
		}
	}
	printf("TEST PASSED\n");
	return 1;
}



__global__ void gpuPrefixScan(int* y, int* x, int* b, int blockCount) {
	__shared__ int scan_array[2 * 128];

	unsigned int t = threadIdx.x;
	unsigned int start = 2 * blockIdx.x * blockDim.x;
	scan_array[t] = x[start + t];
	scan_array[blockDim.x + t] = x[start + blockDim.x + t];

	__syncthreads();

	//reduction step
	int stride = 1;
	int index;
	while(stride <= blockDim.x) {
		index = (threadIdx.x + 1) * stride * 2 - 1;
		if(index < 2 * blockDim.x) {
			scan_array[index] += scan_array[index - stride];
		}
		stride = stride * 2;

		__syncthreads();
	}

	//post scan step
	stride = blockDim.x / 2;
	while(stride > 0) {
		index = (threadIdx.x + 1) * stride * 2 - 1;
		if(index + stride < 2 * blockDim.x) {
			scan_array[index + stride] += scan_array[index];
		}
		stride = stride / 2;
		__syncthreads();
	}
	__syncthreads();

	y[start + t] = scan_array[t];
	y[start + blockDim.x + t] = scan_array[blockDim.x + t];
	b[blockIdx.x] = scan_array[(2 * blockDim.x) - 1];
	__syncthreads();

	//reduction step on block sums
	stride = 1;
	while(stride <= blockCount) {
		index = (threadIdx.x + 1) * stride * 2 - 1;
		if(index < 2 * blockCount) {
			b[index] += b[index - stride];
		}
		stride = stride * 2;

		__syncthreads();
	}

	//post scan step on block sums
	stride = blockCount / 2;
	while(stride > 0) {
		index = (threadIdx.x + 1) * stride * 2 - 1;
		if(index + stride < 2 * blockCount) {
			b[index + stride] += b[index];
		}
		stride = stride / 2;
		__syncthreads();
	}
	__syncthreads();


	__syncthreads();
	if(blockIdx.x != (blockCount - 1)) {
		y[((blockIdx.x + 1) * 2 * blockDim.x) + t] += b[blockIdx.x];
		y[((blockIdx.x + 1) * 2 * blockDim.x) + t + blockDim.x] += b[blockIdx.x];
	}
	__syncthreads();
}

int main() {
	int* A;
	int N = 2048;
	A = (int*) malloc(sizeof(int) * N);
	int init = 1325;
	for(int i = 0; i < N; i++) {
		init = (3125 * init) % 6553;
		A[i] = (init - 1000) % 97;
	}
	printf("Input array has %d elements\n", N);
	dim3 dimBlock(128);
	dim3 dimGrid(ceil((double) N / (2 * dimBlock.x)));
	printf("Using %d thread blocks of %d threads each\n", dimGrid.x, dimBlock.x);

	int* result = (int*) malloc(sizeof(int) * N);
	int* cpuOut = (int*) malloc(sizeof(int) * N);
	int* blockSum = (int*) malloc(sizeof(int) * dimGrid.x * 2);
	int* O = (int*) malloc(sizeof(int) * N);
	O[0] = A[0];

	int* a;
	int* out;
	int* bsum;
	hipMalloc((void**)(&a), sizeof(int) * N);
	hipMalloc((void**)(&out), sizeof(int) * N);
	hipMalloc((void**)(&bsum), sizeof(int) * dimGrid.x * 2);
	hipMemcpy(a, A, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(out, O, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(bsum, blockSum, sizeof(int) * dimGrid.x, hipMemcpyHostToDevice);

	double cpuStartTime;
	double cpuTimeDifference;
	cpuStartTime = (double) clock();
	cpuPrefixScan(cpuOut, A, N);
	cpuTimeDifference = ((double) clock() - cpuStartTime) / CLOCKS_PER_SEC;

	double gpuStartTime;
	double gpuTimeDifference;
	gpuStartTime = (double) clock();
	gpuPrefixScan<<<dimGrid, dimBlock.x>>>(out, a, bsum, dimGrid.x);
	gpuTimeDifference = ((double) clock() - gpuStartTime) / CLOCKS_PER_SEC;

	hipMemcpy(result, out, sizeof(int) * N, hipMemcpyDeviceToHost);

	verify(cpuOut, result, N);
	printf("CPU runtime: %f seconds\n", cpuTimeDifference);
	printf("GPU runtime: %f seconds\n", gpuTimeDifference);

	printf("First 20 elements of the input array: ");
	for(int i = 0; i < 20; i++) {
		printf("%d", A[i]);
		printf("  ");
	}
	printf("\n");

	printf("First 20 elements of the CPU prefix scan array: ");
	for(int i = 0; i < 20; i++) {
		printf("%d", cpuOut[i]);
		printf("  ");
	}
	printf("\n");

	printf("First 20 elements of the GPU prefix scan array: ");
	for(int i = 0; i < 20; i++) {
		printf("%d", result[i]);
		printf("  ");
	}
	printf("\n");

	free(A);
	free(O);
	free(cpuOut);
	free(blockSum);
	hipFree(a);
	hipFree(out);
	hipFree(bsum);
	return 0;
}