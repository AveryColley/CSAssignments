//CS4370
//Parallel Programming for Many-Core GPUs
//Avery Colley
//Meilin Liu
//12/12/24
//Programming Assignment #2 - Tiled Matrix Multiplication
//Compiling command: nvcc Colley_Project2.cu -o matrix

#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

int verify(int *a, int *b, int size) {
	for(int i = 0; i < size; i++) {
		if(a[i] != b[i]) {
			printf("TEST FAILED\n");
			return 0;
		}
	}
	printf("TEST PASSED\n");
	return 1;
}

void cpuMatrixMulti(int *M, int *N, int *P, int width) {
	for(int row = 0; row < width; ++row) {
		for(int col = 0; col < width; ++col) {
			int sum = 0;
			for(int k = 0; k < width; ++k) {
				float a = M[row * width +k];
				float b = N[k * width + col];
				sum += a * b;
			}
			P[row * width + col] = sum;
		}
	}
}

__global__ void gpuMatrixMulti(int *d_M, int *d_N, int *d_P, int width) {
	const int TILE_WIDTH = 8;
	__shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row = by * TILE_WIDTH + ty;
	int col = bx * TILE_WIDTH + tx;
	int pValue = 0;

	for(int m = 0; m < width / TILE_WIDTH; ++m) {
		ds_M[ty][tx] = d_M[row * width + m * TILE_WIDTH + tx];
		ds_N[ty][tx] = d_N[col + (m * TILE_WIDTH + ty) * width];
		__syncthreads();
		for(int k = 0; k < TILE_WIDTH; ++k) {
			pValue += ds_M[ty][k] * ds_N[k][tx];
		}
		__syncthreads();
	}
	d_P[row * width + col] = pValue;
}

int main() {
	int *A;
	int *B;
	int *C;
	int N = 4096;
	A = (int*) malloc(sizeof(int) * N * N);
	B = (int*) malloc(sizeof(int) * N * N);
	C = (int*) malloc(sizeof(int) * N * N);
	int init = 1325;
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			init = 3125 * init % 6553;
			A[(i * N) + j] = (init - 1000) % 6553;
			B[(i * N) + j] = init % 251;
			C[(i * N) + j] = 0;
		}
	}
	printf("Input matrices have %d elements per row\n", N);

	int *a;
	int *b;
	int *c;
	int *result = (int*) malloc(sizeof(int) * N *N);

	hipMalloc((void**)(&a), sizeof(int) * N * N);
	hipMalloc((void**)(&b), sizeof(int) * N * N);
	hipMalloc((void**)(&c), sizeof(int) * N * N);

	hipMemcpy(a, A, sizeof(int) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(b, B, sizeof(int) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(c, C, sizeof(int) * N * N, hipMemcpyHostToDevice);

	const int BLOCK_SIZE = 8;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil((double) N / dimBlock.x), ceil((double) N / dimBlock.y));
	printf("Using %d thread blocks of %dx%d threads each\n", dimGrid.x * dimGrid.y, dimBlock.x, dimBlock.y);

	double cpuStartTime;
	double cpuTimeDifference;
	cpuStartTime = (double) clock();
	cpuMatrixMulti(A, B, C, N);
	cpuTimeDifference = ((double) clock() - cpuStartTime) / CLOCKS_PER_SEC;

	double gpuStartTime;
	double gpuTimeDifference;
	gpuStartTime = (double) clock();
	gpuMatrixMulti<<<dimGrid, dimBlock>>>(a, b, c, N);
	gpuTimeDifference = ((double) clock() - gpuStartTime) / CLOCKS_PER_SEC;

	hipMemcpy(result, c, sizeof(int) * N * N, hipMemcpyDeviceToHost);
	verify(C, result, N * N);
	printf("CPU runtime: %f seconds\n", cpuTimeDifference);
	printf("GPU runtime: %f seconds\n", gpuTimeDifference);

	printf("First row of CPU result matrix: [");
	for(int i = 0; i < N; i++) {
		printf("%d", C[i]);
		printf(" ");
	}
	printf("]\n");

	printf("First row of GPU result matrix: [");
	for(int i = 0; i < N; i++) {
		printf("%d", result[i]);
		printf(" ");
	}
	printf("]\n");

	free(A);
	free(B);
	free(C);
	free(result);
	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}