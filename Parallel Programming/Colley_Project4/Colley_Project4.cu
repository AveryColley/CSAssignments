//CS4370
//Parallel Programming for Many-Core GPUs
//Avery Colley
//Meilin Liu
//12/6/24
//Programming Assignment #4 -- Histogram
//Compiling command: nvcc Colley_Project4.cu -o histogram -arch=sm_30

#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

int verify(int *a, int *b, int size) {
	for(int i = 0; i < size; i++) {
		if(a[i] != b[i]) {
			printf("TEST FAILED\n");
			return 0;
		}
	}
	printf("TEST PASSED\n");
	return 1;
}

void compute_histogram(int *input, int size, int* output) {
		for(int i = 0; i < 256; i++) {
			output[i] = 0;
		}
		for(int i = 0; i < size; i++) {
			output[input[i]]++;
		}
}

__global__ void atomic_histogram_kernel(unsigned int *buffer, long size, int *histo) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while (i < size) {
		atomicAdd(&(histo[buffer[i]]), 1);
		i += stride;
	}
}

__global__ void shared_histogram_kernel(unsigned int *buffer, long size, int *histo) {
	__shared__ unsigned int histo_private[256];
	if(threadIdx.x < 256) {
		histo_private[threadIdx.x] = 0;
	}
	__syncthreads();

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while( i < size) {
		atomicAdd(&(histo_private[buffer[i]]), 1);
		i += stride;
	}

	__syncthreads();
	if(threadIdx.x < 256) {
		atomicAdd(&(histo[threadIdx.x]), histo_private[threadIdx.x]);
	}
}

int main() {
	int *A;
	int N = 131072;
	A = (int*) malloc(sizeof(int) * N );
	int init = 1325;
	for(int i = 0; i < N; i++) {
		init = 3125 * init % 65537;
		A[i] = init % 256;
	}
	printf("Input array has %d elements\n", N);

	int* H;
	H = (int*) malloc(sizeof(int) * 256);
	for(int i = 0; i < 256; i++) {
		H[i] = 0;
	}

	int *output = (int*) malloc(sizeof(int) * 256);


	unsigned int* dev_a;
	int* histo;
	int* result = (int*) malloc(sizeof(int) * 256);
	hipMalloc((void**)(&dev_a), sizeof(int) * N);
	hipMalloc((void**)(&histo), sizeof(int) * 256);
	hipMemcpy(dev_a, A, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(histo, H, sizeof(int) * 256, hipMemcpyHostToDevice);
	const int BLOCK_SIZE = 512;
	int blockCount = ceil (N / double(BLOCK_SIZE));
	printf("Using %d thread blocks of %d threads each\n", blockCount, BLOCK_SIZE);



	double cpuStartTime;
	double cpuTimeDifference;
	cpuStartTime = (double) clock();
	compute_histogram(A, N, output);
	cpuTimeDifference = ((double) clock() - cpuStartTime) / CLOCKS_PER_SEC;

	double gpuStartTime;
	double gpuTimeDifference;
	gpuStartTime = (double) clock();
	shared_histogram_kernel<<<blockCount, BLOCK_SIZE>>>(dev_a, N, histo);
	gpuTimeDifference = ((double) clock() - gpuStartTime) / CLOCKS_PER_SEC;

	hipMemcpy(result, histo, sizeof(int) * 256, hipMemcpyDeviceToHost);
	verify(result, output, 256);
	printf("CPU runtime: %f seconds\n", cpuTimeDifference);
	printf("GPU runtime: %f seconds\n", gpuTimeDifference);

	printf("First 10 elements of the input array: ");
	for(int i = 0; i < 10; i++) {
		printf("%d", A[i]);
		printf("  ");
	}
	printf("\n");

	printf("First 10 elements of the CPU histogram: ");
	for(int i = 0; i < 10; i++) {
		printf("%d", output[i]);
		printf("  ");
	}
	printf("\n");

	printf("First 10 elements of the GPU histogram: ");
	for(int i = 0; i < 10; i++) {
		printf("%d", result[i]);
		printf("  ");
	}
	printf("\n");

	free(A);
	free(result);
	free(output);
	hipFree(dev_a);
	hipFree(histo);
	return 0;
}